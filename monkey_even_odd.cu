#include "hip/hip_runtime.h"
#pragma once

#ifdef __INTELLISENSE__

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#define __HIPCC__

#include <hip/device_functions.h>

#endif

//#include <hip/hip_runtime.h>
// CUDA runtime
//#include "hip/hip_runtime.h"
//#include ""

// nvcc does not seem to like variadic macros, so we have to define
// one for each kernel parameter list:
#ifdef __HIPCC__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "monkey_even_odd.h"
//#include <thrust/sort.h>

// Now launch your kernel using the appropriate macro:
//kernel KERNEL_ARGS2(dim3(nBlockCount), dim3(nThreadCount)) (param1);
//https://stackoverflow.com/a/27992604/5241172

/*
nvcc -O3 -arch=sm_30 -o even_odd_cuda_monkey monkey_even_odd.cu
*/

unsigned int print2Smallest(unsigned int *arr, unsigned int arr_size)
{
  unsigned int i, first, second;

  /* There should be atleast two elements */
  if (arr_size < 2)
  {
    printf(" Invalid Input ");
    return 0;
  }

  first = second = UINT_MAX;
  for (i = 0; i < arr_size ; i ++)
  {
    /* If current element is smaller than first 
       then update both first and second */
    if (arr[i] < first)
    {
      second = first;
      first = arr[i];
    }

    /* If arr[i] is in between first and second 
       then update second  */
    else if (arr[i] < second && arr[i] != first)
      second = arr[i];
  }
  
  if (second == UINT_MAX)
    return first;
  else 
    return second;
}


__global__ 
void monkey(unsigned int *tot, unsigned long long int extra, unsigned int *the_solutions, unsigned int *found, unsigned int sailors, unsigned int monkeys, unsigned int n)
{
  if (found[0] == 0){

    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=blockDim.x*gridDim.x){

	  unsigned char j;
	  const unsigned long long int rExtra = extra;
	  unsigned int rCoconuts = tot[i] + rExtra;
	  const unsigned int rSailors = sailors;
	  const unsigned int rMonkeys = monkeys;

      // Go through the number of sailors
      for (j=0; j<rSailors;j++){
        // One for each monkey
		  rCoconuts -= rMonkeys;
        if (rCoconuts % rSailors != 0){
          break;
        }     
		rCoconuts -= rCoconuts/ rSailors;
      }
      if (rCoconuts % rSailors == 0){
        found[0] = 1;
        the_solutions[i] = i;
/*
        printf("i=%d",i);
        for (j=0;j<6;j++){
          printf("tot[%u]+extra(=%llu)=%d\n", i+j, extra,tot[i+j]+extra);
        }


        //printf("extra=%llu\n", extra);

        // test om der kopieres eller de begge bliver talt ned. de bliver kopieret, dvs. tot ikke skal reinitialiseres.
        // og det skal coconuts heller ikke
        printf("coconuts[%d]=%llu, tot[%d]+extra = %llu\n", i, coconuts[i], i, tot[i]+extra);
*/
      }
    }
  }
}

// Main method
int main()
{

  clock_t start, diff;
  
  // Size of array.
  unsigned int SIZE = (unsigned int)pow(2, 21);// 65536;// 65025;// / 2;//(unsigned int)pow(2,25);
  //unsigned int SIZE = 1024;// (unsigned int)pow(2, 20);

  /*
  // Possibly used to automate the SIZE value. For now, I found 2^21 to give good results 
  size_t availableMemory, totalMemory, usedMemory;

  hipMemGetInfo(&availableMemory, &totalMemory);
  usedMemory = totalMemory - availableMemory;
  */


  int blockSize;   // The launch configurator returned block size 
  int minGridSize; // The minimum grid size needed to achieve the 
				   // maximum occupancy for a full device launch 
  int gridSize;    // The actual grid size needed, based on input size 


  // Sailors and monkeys
  unsigned char monkeys = 1;
  unsigned char max_sailors = 9;

  // CPU memory pointers
  unsigned long long int *h_coc, da_solu=1;
  unsigned int *h_found, *h_solutions, *h_tot;

  // GPU memory pointers
  unsigned long long int extra = 0;
  unsigned int *d_found, *d_solutions, *d_tot;

  // Allocate the space, CPU
  h_coc = (unsigned long long int *)malloc(SIZE*sizeof(unsigned long long int));
  hipHostAlloc((void**)&h_solutions, SIZE*sizeof(unsigned int), hipHostMallocDefault);
  h_found = (unsigned int *)malloc(1*sizeof(unsigned int));
  h_tot = (unsigned int *)malloc(SIZE*sizeof(unsigned int));
  
  // Choose to run on secondary GPU
  hipSetDevice(1);

  // Allocate the space, GPU
  //hipMalloc(&d_coc, SIZE*sizeof(unsigned long long int));
  hipMalloc(&d_found, 1*sizeof(unsigned int));
  hipMalloc(&d_solutions, SIZE*sizeof(unsigned int));
  hipMalloc(&d_tot, SIZE*sizeof(unsigned int));

  //cudamemset can be used for initializing data (say, all zeros). 10 times faster than hipMemcpy zero array because it is done on the gpu directly.
  //hipMemset(d_solutions, 0, SIZE*sizeof(unsigned int));

  // Initialise the data
  unsigned int i, j=0;
  
  if (monkeys%2){
    //printf("odd\n");
    //solution will be odd
    i=1;
  }
  else{
    //printf("even\n");
    //solution will be even
    i=2;
  }

  while (j < SIZE)
  {
    //h_coc[j] = i;
    h_tot[j] = i;
    j++;
    i+=2;
  }

/*
  for (i=0;i<6;i++){
    printf("h_coc[%u]=%llu\n", i, h_coc[i]);
  }

  printf("h_coc[SIZE-1]=%llu\n", h_coc[SIZE-1]);

*/
  /*
  for (i=0;i<6;i++){
    printf("h_tot[%u]=%d\n", i, h_tot[i]);
  }

  printf("h_tot[SIZE-1]=%d\n", h_tot[SIZE-1]);
*/


  /*
  Transfer this to gpu*/
 // hipMemcpy(d_coc, h_coc, SIZE*sizeof(unsigned long long int), hipMemcpyHostToDevice);
  hipMemcpy(d_tot, h_tot, SIZE*sizeof(unsigned int), hipMemcpyHostToDevice);

  //cudamemset can be used for initializing data (say, all zeros). 10 times faster than hipMemcpy zero array because it is done on the gpu directly.
  hipMemset(d_solutions, 0, SIZE*sizeof(unsigned int));

  // Get some qualified guess as to how to choose gridsize and blocksize
  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
	  monkey, 0, SIZE);
  // Round up according to array size 
  gridSize = (SIZE + blockSize - 1) / blockSize;


  // Start timer
  start = clock();

  // Run the loop 
  for (unsigned char sailors=2; sailors<max_sailors+1;sailors++){

    printf("Running %u sailors, %u monkeys\n", sailors, monkeys);

    // Send back that we want to look for a new solution
    h_found[0] = 0; 
    hipMemset(d_found, 0, 1*sizeof(unsigned int));

    // Assume that result for 5 sailors is larger than for 4 sailors and so on.. 
    //extra += h_tot[da_solu]+1;
    //printf("extra = %llu\n", extra);
    // Run this loop until a solution is found for this sailor & monkey combination
    

    while (h_found[0] == 0){
/*
      for (i=0;i<6;i++){
      printf("h_tot[%u]=%llu\n", i, h_tot[i]+extra);
      }

      printf("h_tot[SIZE-1]=%llu\n", h_tot[SIZE-1]+extra);
  */    

      // Calling kernel (gridsize, blocksize found above)
	  monkey KERNEL_ARGS2(gridSize, blockSize)(d_tot, extra, d_solutions, d_found, sailors, monkeys, SIZE);

      // Copy back result (Device to Host). 
      hipMemcpy(h_found, d_found, 1*sizeof(unsigned int), hipMemcpyDeviceToHost);

      if (h_found[0] == 1){

        //printf("extra = %llu\n", extra);

        // Copy back result (Device to Host). This is pinned memory so +6 Gb/s
        hipMemcpy(h_solutions, d_solutions, SIZE*sizeof(unsigned int), hipMemcpyDeviceToHost);
        
        //hipMemcpyAsync(h_solutions, d_solutions, SIZE*sizeof(unsigned int), hipMemcpyDeviceToHost, 0);
        //hipDeviceSynchronize();
        
        // Get second smallest in solutions array and recast
        // possibly do this on gpu as well
        da_solu = (unsigned long long int) print2Smallest(h_solutions, SIZE); 

        printf("Solution: %llu coconuts to begin with\n\n", h_tot[da_solu]+extra);

        if (sailors != max_sailors){
          // Set solution array to zero again
          hipMemset(d_solutions, 0, SIZE*sizeof(unsigned int));   
        }
      }
      else{
        // should always be equal amount
        extra += 2*SIZE; // size is even times 2 since we only use hver anden
        //printf("."); 
      }
    }
    extra += h_tot[da_solu]+1;
  }
  
  // watch -n 0.5 "nvidia-settings -q GPUUtilization -q useddedicatedgpumemory"

  // Print execution time
  diff = clock() - start;
  double totalt = (double)diff/CLOCKS_PER_SEC;
  printf("Totalt: %f s\n", totalt);

  // Free the allocated memory
  free(h_coc);
  free(h_found);
  free(h_tot);
  //free(h_solutions);
  // Pinned memory needs to be released with the command
  hipHostFree(h_solutions);

  // Free GPU memory
  //hipFree(d_coc);  
  hipFree(d_tot);
  hipFree(d_found);
  hipFree(d_solutions);

  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits
  hipDeviceReset();

  return 0;
}