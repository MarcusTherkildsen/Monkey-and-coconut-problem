
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
//#include <thrust/sort.h>

/*
nvcc -O3 -arch=sm_30 -o even_odd_cuda_monkey monkey_even_odd.cu
*/

unsigned int print2Smallest(unsigned int *arr, unsigned int arr_size)
{
  unsigned int i, first, second;

  /* There should be atleast two elements */
  if (arr_size < 2)
  {
    printf(" Invalid Input ");
    return 0;
  }

  first = second = UINT_MAX;
  for (i = 0; i < arr_size ; i ++)
  {
    /* If current element is smaller than first 
       then update both first and second */
    if (arr[i] < first)
    {
      second = first;
      first = arr[i];
    }

    /* If arr[i] is in between first and second 
       then update second  */
    else if (arr[i] < second && arr[i] != first)
      second = arr[i];
  }
  
  if (second == UINT_MAX)
    return first;
  else 
    return second;
}


__global__
void monkey(unsigned long long int *coconuts, unsigned int *tot, unsigned long long int extra, unsigned int *the_solutions, unsigned int *found, unsigned int sailors, unsigned int monkeys, unsigned int n)
{
  if (found[0] == 0){

    unsigned int j;
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=blockDim.x*gridDim.x){

      coconuts[i] = tot[i]+extra;

      // Go through the number of sailors
      for (j=0; j<sailors;j++){
        // One for each monkey
        coconuts[i] -= monkeys;      
        if (coconuts[i] % sailors != 0){
          break;
        }     
        coconuts[i] -= coconuts[i]/sailors;   
      }
      if (coconuts[i] % sailors == 0){ 
        found[0] = 1;
        the_solutions[i] = i;
/*
        printf("i=%d",i);
        for (j=0;j<6;j++){
          printf("tot[%u]+extra(=%llu)=%d\n", i+j, extra,tot[i+j]+extra);
        }


        //printf("extra=%llu\n", extra);

        // test om der kopieres eller de begge bliver talt ned. de bliver kopieret, dvs. tot ikke skal reinitialiseres.
        // og det skal coconuts heller ikke
        printf("coconuts[%d]=%llu, tot[%d]+extra = %llu\n", i, coconuts[i], i, tot[i]+extra);
*/
      }
    }
  }
}

// Main method
int main()
{

  clock_t start, diff;
  
  // Size of array.
  unsigned int SIZE = pow(2,25);

  // Sailors and monkeys
  unsigned int monkeys = 1;
  unsigned int max_sailors = 9;

  // CPU memory pointers
  unsigned long long int *h_coc, da_solu=1;
  unsigned int *h_found, *h_solutions, *h_tot;

  // GPU memory pointers
  unsigned long long int *d_coc, extra = 0;
  unsigned int *d_found, *d_solutions, *d_tot;

  // Allocate the space, CPU
  h_coc = (unsigned long long int *)malloc(SIZE*sizeof(unsigned long long int));
  hipHostAlloc((void**)&h_solutions, SIZE*sizeof(unsigned int), hipHostMallocDefault);
  h_found = (unsigned int *)malloc(1*sizeof(unsigned int));
  h_tot = (unsigned int *)malloc(SIZE*sizeof(unsigned int));
  
  // Choose to run on secondary GPU
  //cudaSetDevice(1);

  // Allocate the space, GPU
  hipMalloc(&d_coc, SIZE*sizeof(unsigned long long int));
  hipMalloc(&d_found, 1*sizeof(unsigned int));
  hipMalloc(&d_solutions, SIZE*sizeof(unsigned int));
  hipMalloc(&d_tot, SIZE*sizeof(unsigned int));

  //cudamemset can be used for initializing data (say, all zeros). 10 times faster than cudaMemcpy zero array because it is done on the gpu directly.
  //cudaMemset(d_solutions, 0, SIZE*sizeof(unsigned int));

  // Initialise the data
  unsigned int i, j=0;
  
  if (monkeys%2){
    //printf("odd\n");
    //solution will be odd
    i=1;
  }
  else{
    //printf("even\n");
    //solution will be even
    i=2;
  }

  while (j < SIZE)
  {
    //h_coc[j] = i;
    h_tot[j] = i;
    j++;
    i=i+2;
  }

/*
  for (i=0;i<6;i++){
    printf("h_coc[%u]=%llu\n", i, h_coc[i]);
  }

  printf("h_coc[SIZE-1]=%llu\n", h_coc[SIZE-1]);

*/
  /*
  for (i=0;i<6;i++){
    printf("h_tot[%u]=%d\n", i, h_tot[i]);
  }

  printf("h_tot[SIZE-1]=%d\n", h_tot[SIZE-1]);
*/


  /*
  Transfer this to gpu*/
 // cudaMemcpy(d_coc, h_coc, SIZE*sizeof(unsigned long long int), cudaMemcpyHostToDevice);
  hipMemcpy(d_tot, h_tot, SIZE*sizeof(unsigned int), hipMemcpyHostToDevice);

  //cudamemset can be used for initializing data (say, all zeros). 10 times faster than cudaMemcpy zero array because it is done on the gpu directly.
  hipMemset(d_solutions, 0, SIZE*sizeof(unsigned int));

  // Start timer
  start = clock();

  // Run the loop 
  for (unsigned int sailors=2; sailors<max_sailors+1;sailors++){

    printf("Running %u sailors, %u monkeys\n", sailors, monkeys);

    // Send back that we want to look for a new solution
    h_found[0] = 0; 
    hipMemset(d_found, 0, 1*sizeof(unsigned int));

    // Assume that result for 5 sailors is larger than for 4 sailors and so on.. 
    //extra += h_tot[da_solu]+1;
    //printf("extra = %llu\n", extra);
    // Run this loop until a solution is found for this sailor & monkey combination
    

    while (h_found[0] == 0){
/*
      for (i=0;i<6;i++){
      printf("h_tot[%u]=%llu\n", i, h_tot[i]+extra);
      }

      printf("h_tot[SIZE-1]=%llu\n", h_tot[SIZE-1]+extra);
  */    

      // Calling kernel (gridsize, blocksize)
      monkey<<<(SIZE + 255) / 256, 256>>>(d_coc, d_tot, extra, d_solutions, d_found, sailors, monkeys, SIZE);

      // Copy back result (Device to Host). 
      hipMemcpy(h_found, d_found, 1*sizeof(unsigned int), hipMemcpyDeviceToHost);

      if (h_found[0] == 1){

        //printf("extra = %llu\n", extra);

        // Copy back result (Device to Host). This is pinned memory so +6 Gb/s
        hipMemcpy(h_solutions, d_solutions, SIZE*sizeof(unsigned int), hipMemcpyDeviceToHost);
        
        //cudaMemcpyAsync(h_solutions, d_solutions, SIZE*sizeof(unsigned int), cudaMemcpyDeviceToHost, 0);
        //cudaDeviceSynchronize();
        
        // Get second smallest in solutions array and recast
        // possibly do this on gpu as well
        da_solu = (unsigned long long int) print2Smallest(h_solutions, SIZE); 

        printf("Solution: %llu coconuts to begin with\n\n", h_tot[da_solu]+extra);

        if (sailors != max_sailors){
          // Set solution array to zero again
          hipMemset(d_solutions, 0, SIZE*sizeof(unsigned int));
          
        }
      }
      else{
        // should always be equal amount
        extra +=2*SIZE; // size is even times 2 since we only use hver anden
        //printf("."); 
      }
    }
    extra += h_tot[da_solu]+1;
  }
  
  // watch -n 0.5 "nvidia-settings -q GPUUtilization -q useddedicatedgpumemory"

  // Print execution time
  diff = clock() - start;
  double totalt = (double)diff/CLOCKS_PER_SEC;
  printf("Totalt: %f s\n", totalt);

  // Free the allocated memory
  free(h_coc);
  free(h_found);
  free(h_tot);
  //free(h_solutions);
  // Pinned memory needs to be released with the command
  hipHostFree(h_solutions);

  // Free GPU memory
  hipFree(d_coc);  
  hipFree(d_tot);
  hipFree(d_found);
  hipFree(d_solutions);

  // cudaDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling cudaDeviceReset causes all profile data to be
  // flushed before the application exits
  hipDeviceReset();

  return 0;
}