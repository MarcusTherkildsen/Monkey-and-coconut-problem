
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

/*
nvcc -O3 -arch=sm_30 -o cuda_monkey monkey.cu
*/

unsigned int print2Smallest(unsigned int *arr, unsigned int arr_size)
{
  unsigned int i, first, second;

  /* There should be atleast two elements */
  if (arr_size < 2)
  {
    printf(" Invalid Input ");
    return 0;
  }

		// Error was here, before we had INT_MAX which is too low for >9 sailors
  first = second = UINT_MAX;
  for (i = 0; i < arr_size ; i ++)
  {
    /* If current element is smaller than first 
       then update both first and second */
    if (arr[i] < first)
    {
        second = first;
        first = arr[i];
    }

    /* If arr[i] is in between first and second 
       then update second  */
    else if (arr[i] < second && arr[i] != first)
      second = arr[i];
  }
  
  if (second == UINT_MAX)
  	return first;
  else 
		return second;
}


__global__
void monkey(unsigned long long int *coconuts, unsigned long long int extra, unsigned int *the_solutions, unsigned int *found, unsigned int sailors, unsigned int monkeys, unsigned int n)
{
	if (found[0] == 0){

		unsigned int j;
		for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i<n; i+=blockDim.x*gridDim.x){

			coconuts[i] = i + extra;

		  // Go through the number of sailors
		  for (j=0; j<sailors;j++){
	      // One for each monkey
	      coconuts[i] -= monkeys;      
	      if (coconuts[i] % sailors != 0){
	        break;
	      }			
	      coconuts[i] -= coconuts[i]/sailors;	  
		  }
		  if (coconuts[i] % sailors == 0){ 
	    	found[0] = 1;
	    	the_solutions[i] = i;   
		  }
		}
	}
}

// Main method
int main()
{

	clock_t start, diff;
	
	// Size of array.
	unsigned int SIZE = pow(2,25);

	// CPU memory pointers
	unsigned long long int *h_coc, da_solu=0;
	unsigned int *h_found, *h_solutions;

	// GPU memory pointers
	unsigned long long int *d_coc, extra = 0;
	unsigned int *d_found, *d_solutions;

	// Allocate the space, CPU
	h_coc = (unsigned long long int *)malloc(SIZE*sizeof(unsigned long long int));
	h_solutions = (unsigned int *)malloc(SIZE*sizeof(unsigned int));
	h_found = (unsigned int *)malloc(1*sizeof(unsigned int));
	
	// Choose to run on secondary GPU
	hipSetDevice(1);

	// Allocate the space, GPU
	hipMalloc(&d_coc, SIZE*sizeof(unsigned long long int));
	hipMalloc(&d_found, 1*sizeof(unsigned int));
	hipMalloc(&d_solutions, SIZE*sizeof(unsigned int));

	//cudamemset can be used for initializing data (say, all zeros). Basically same speed as cudaMemcpy but simpler.
	hipMemset(d_solutions, 0, SIZE*sizeof(unsigned int));

	// Start timer
	start = clock();

	unsigned int monkeys = 1;

	// Run the loop	
	for (unsigned int sailors=1; sailors<11;sailors++){

		printf("Running %u sailors, %u monkeys", sailors, monkeys);

		// Send back that we want to look for a new solution
	  h_found[0] = 0; 
	  hipMemset(d_found, 0, 1*sizeof(unsigned int));

	  // Assume that result for 5 sailors is larger than for 4 sailors and so on.. 
	  extra = da_solu+extra;

	  // Run this loop until a solution is found for this sailor & monkey combination
		while (h_found[0] == 0){

			// Calling kernel (gridsize, blocksize)
			monkey<<<(SIZE + 255) / 256, 256>>>(d_coc, extra, d_solutions, d_found, sailors, monkeys, SIZE);

			// Copy back result (Device to Host)
			hipMemcpy(h_found, d_found, 1*sizeof(unsigned int), hipMemcpyDeviceToHost);

			if (h_found[0] == 1){

				// Copy back result (Device to Host)
				hipMemcpy(h_solutions, d_solutions, SIZE*sizeof(unsigned int), hipMemcpyDeviceToHost);

			  // Get second smallest in solutions array and recast
			  da_solu = (unsigned long long int) print2Smallest(h_solutions, SIZE); 

			  printf("\nSolution: %llu coconuts to begin with\n\n", da_solu+extra);

			  // Set solution array to zero again
				hipMemset(d_solutions, 0, SIZE*sizeof(unsigned int));

			}
			else{
				extra +=SIZE; 
				//printf(".");
			}
		}
	}
	
	// watch -n0.1 "nvidia-settings -q GPUUtilization -q useddedicatedgpumemory"

	// Print execution time
	diff = clock() - start;
  double totalt = (double)diff/CLOCKS_PER_SEC;
  printf("Totalt: %f s\n", totalt);

	// Free the allocated memory
	free(h_coc);
	free(h_found);
	free(h_solutions);

	// Free GPU memory
	hipFree(d_coc);	
	hipFree(d_found);
	hipFree(d_solutions);

	// cudaDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling cudaDeviceReset causes all profile data to be
  // flushed before the application exits
  hipDeviceReset();

	return 0;
}